

#include "cuda_computing.cuh"

#define THREADS_PER_BLOCK 128
#define BODIES_PER_THREAD 4 //only mutiples of 2

namespace Device {
	// CUDA global constants
	__device__ __constant__
		int NTHREADS;

////////////////////////////////////////////////////////////////////////////////////////////////////
// initializes device, detects hardware, number of threads per block
////////////////////////////////////////////////////////////////////////////////////////////////////
bool
Cuda_Computing::initDevice() {
	//check execution environment
	int deviceCount = 0;
	int device_handle = 0;
	errorCheckCuda(hipGetDeviceCount(&deviceCount));

	if (0 == deviceCount) {
		std::cerr << "initDevice() : No CUDA device found." << std::endl;
		return false;
	}

	if (deviceCount > 1) {
		std::cerr << "initDevice() : Multiple CUDA devices found. Using first one." << std::endl;
	}

	// set the device
	errorCheckCuda(hipSetDevice(device_handle));

	hipDeviceProp_t device_props;
	errorCheckCuda(hipGetDeviceProperties(&device_props, device_handle));
	//std::cerr << "Max CC: " << device_props.major << "   Min CC: " << device_props.minor << std::endl;

	// determine thread layout
	// num of threads on 1 block, thread layout per block
	blockSize = dim3(THREADS_PER_BLOCK, 1, 1);
	int numBlocks = N / THREADS_PER_BLOCK;
	if (0 != N % blockSize.x) numBlocks++;
	// number of blocks, block layout on grid
	gridSize = dim3(numBlocks, 1, 1);

	std::cerr << "num blocks = " << gridSize.x << " :: "
		<< "threads per Block = " << blockSize.x << std::endl;

	int nTh = THREADS_PER_BLOCK;

	errorCheckCuda(hipMemcpyToSymbol(HIP_SYMBOL(Device::NTHREADS), &blockSize.x, sizeof(int), 0, hipMemcpyHostToDevice));
	return true;
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// kernel entry point
////////////////////////////////////////////////////////////////////////////////////////////////////
float
Cuda_Computing::compute() {
	float time;
	hipEvent_t start, stop;
	errorCheckCuda(hipEventCreate(&start));
	errorCheckCuda(hipEventCreate(&stop));
	errorCheckCuda(hipEventRecord(start, 0));

	//CALL CUDA COMPUTATION

	//errorCheckCuda(hipPeekAtLastError());
	errorCheckCuda(hipDeviceSynchronize());
	errorCheckCuda(hipEventRecord(stop, 0));
	errorCheckCuda(hipEventSynchronize(stop));
	errorCheckCuda(hipEventElapsedTime(&time, start, stop));

	return time;
}
